#include <hipDNN.h>
#include <cassert>
#include <cstdlib>
#include <iostream>

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }




int main(int argc, const char* argv[]) {

	hipdnnHandle_t cudnn;
	hipdnnCreate(&cudnn);

	hipdnnTensorDescriptor_t input_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
				/*format=*/HIPDNN_TENSOR_NHWC,
				/*dataType=*/HIPDNN_DATA_FLOAT,
				/*batch_size=*/S_batch_size,
				/*channels=*/S_channels,
				/*image_height=*/S_input_height,
				/*image_width=*/S_input_width));

	hipdnnFilterDescriptor_t kernel_descriptor;
	checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
	checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
				/*dataType=*/HIPDNN_DATA_FLOAT,
				/*format=*/HIPDNN_TENSOR_NCHW,
				/*out_channels=*/S_kernels,
				/*in_channels=*/S_channels,
				/*kernel_height=*/S_kernel_height,
				/*kernel_width=*/S_kernel_width));

	hipdnnConvolutionDescriptor_t convolution_descriptor;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
				/*pad_height=*/S_padding_height,
				/*pad_width=*/S_padding_width,
				/*vertical_stride=*/S_vertical_stride,
				/*horizontal_stride=*/S_horizontal_stride,
				/*dilation_height=*/S_dilation_height,
				/*dilation_width=*/S_dilation_width,
				/*mode=*/HIPDNN_CROSS_CORRELATION,
				/*computeType=*/HIPDNN_DATA_FLOAT));

	int batch_size_output{0}, channels{0}, height{0}, width{0};
	checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
				input_descriptor,
				kernel_descriptor,
				&batch_size_output,
				&channels,
				&height,
				&width));

	assert(S_batch_size == batch_size_output && channels == S_kernels);

	hipdnnTensorDescriptor_t output_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
				/*format=*/HIPDNN_TENSOR_NHWC,
				/*dataType=*/HIPDNN_DATA_FLOAT,
				/*batch_size=*/S_batch_size,
				/*channels=*/channels,
				/*image_height=*/height,
				/*image_width=*/width));

	hipdnnConvolutionFwdAlgo_t convolution_algorithm;
	checkCUDNN(
			hipdnnGetConvolutionForwardAlgorithm(cudnn,
				input_descriptor,
				kernel_descriptor,
				convolution_descriptor,
				output_descriptor,
				//HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
				HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE,
				/*memoryLimitInBytes=*/0,
				&convolution_algorithm));

	//std::cout << convolution_algorithm << "   ";

	float* d_input{nullptr};
	hipMalloc(&d_input, S_batch_size * S_channels * S_input_height * S_input_width * sizeof(float));


	float* d_output{nullptr};
	hipMalloc(&d_output, S_batch_size * channels * height * width * sizeof(float));

	float* d_kernel{nullptr};
	hipMalloc(&d_kernel, S_kernels * S_channels * S_kernel_height * S_kernel_width);

	const float alpha = 1.0f, beta = 0.0f;

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	checkCUDNN(hipdnnConvolutionForward(cudnn,
				&alpha,
				input_descriptor,
				d_input,
				kernel_descriptor,
				d_kernel,
				convolution_descriptor,
				convolution_algorithm,
				NULL,
				0,
				&beta,
				output_descriptor,
				d_output));

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	//std::cerr << "time to conv: " << time << " ms" << std::endl;
	std::cout << "execution time: " << time << std::endl;


	hipFree(d_kernel);
	hipFree(d_input);
	hipFree(d_output);
	//  hipFree(d_workspace);

	hipdnnDestroyTensorDescriptor(input_descriptor);
	hipdnnDestroyTensorDescriptor(output_descriptor);
	hipdnnDestroyFilterDescriptor(kernel_descriptor);
	hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

	hipdnnDestroy(cudnn);
}
